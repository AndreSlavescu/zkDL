#include "fr-tensor.cuh"
#include "g1-tensor.cuh"
#include "proof.cuh"
#include <iostream>
#include <iomanip>
#include <random>
#include "timer.hpp"

using namespace std;

vector<Fr_t> random_vec(uint len)
{
    std::random_device rd;
    std::mt19937 mt(rd());
    std::uniform_int_distribution<unsigned int> dist(0, UINT_MAX);
    vector<Fr_t> out(len);
    for (uint i = 0; i < len; ++ i) out[i] = {dist(mt), dist(mt), dist(mt), dist(mt), dist(mt), dist(mt), dist(mt), 0};
    return out;
}

int main(int argc, char *argv[])
{
	uint log_m = stoi(argv[1]);
    uint log_n = stoi(argv[2]);
	uint log_p = stoi(argv[3]);
    uint log_nbits = stoi(argv[4]);

    uint m = 1 << log_m;
    uint n = 1 << log_n;
    uint p = 1 << log_p;
    uint nbits = 1 << log_nbits;

	Fr_t* cpu_data_A = new Fr_t[m * n];
	for (uint i = 0; i < m; ++ i)
	{
        for (uint j = 0; j < n; ++ j)
        {
            cpu_data_A[i * n + j] = {0, 0, 0, i, 0, 0, 0, j};
        }
		
	}

    Fr_t* cpu_data_B = new Fr_t[n * p];
    for (uint i = 0; i < n; ++ i)
	{
        for (uint j = 0; j < p; ++ j)
        {
            cpu_data_B[i * p + j] = {0, 0, 0, i, 0, 0, 0, j};
        }
	}

    FrTensor A(m * n, cpu_data_A);
    FrTensor B(n * p, cpu_data_B);

    auto u_m = random_vec(log_m);
    auto u_n = random_vec(log_n);
    auto u_p = random_vec(log_p);

    vector<Fr_t> u_A;
    u_A.insert(u_A.end(), u_n.begin(), u_n.end());
    u_A.insert(u_A.end(), u_m.begin(), u_m.end());
    vector<Fr_t> u_B;
    u_B.insert(u_B.end(), u_p.begin(), u_p.end());
    u_B.insert(u_B.end(), u_n.begin(), u_n.end());


    Timer timer;
    timer.start();
    auto a = A.partial_me(u_m, n);
    auto b = B.partial_me(u_p, 1);
    timer.stop();
    cout << timer.getTotalTime() << endl;
    timer.reset();

    timer.start();
    auto proof = inner_product_sumcheck(a, b, u_n);
    timer.stop();
    cout << timer.getTotalTime() << endl;
    timer.reset();

    timer.start();
    auto y_A = A(u_A);
    auto y_B = B(u_B);
    timer.stop();
    cout << timer.getTotalTime() << endl;
    // cout << y_A << "\t" << a(u_n) << "\t" << proof[proof.size() - 2] << endl;
    // cout << y_B << "\t" << b(u_n) << "\t" << proof[proof.size() - 1] << endl;
    timer.reset();

    Fr_t* cpu_data_C = new Fr_t[m * p];
	for (uint i = 0; i < m; ++ i)
	{
        for (uint j = 0; j < p; ++ j)
        {
            cpu_data_C[i * n + j] = {0, 0, 0, i, 0, 0, 0, j};
        }
		
	}

    Fr_t* cpu_data_D = new Fr_t[m * p];
    for (uint i = 0; i < m; ++ i)
	{
        for (uint j = 0; j < p; ++ j)
        {
            cpu_data_D[i * p + j] = {0, 0, 0, i, 0, 0, 0, j};
        }
	}

    auto u_ip = random_vec(log_m + log_p);
    auto v_ip = random_vec(log_m + log_p);

    FrTensor C(m * p, cpu_data_C);
    FrTensor D(m * p, cpu_data_D);

    timer.start();
    auto ip_proof = hadamard_product_sumcheck(C, D, u_ip, v_ip);
    timer.stop();
    cout << timer.getTotalTime() << endl;
    timer.reset();

    Fr_t* cpu_data_BD = new Fr_t[m * p * nbits];
	for (uint i = 0; i < m * p * nbits; ++ i)
	{
        if (i % 2) cpu_data_BD[i] = {0, 0, 0, 0, 0, 0, 0, 0};
        else cpu_data_BD[i] = {4294967294, 1, 215042, 1485092858, 3971764213, 2576109551, 2898593135, 405057881};
	}
    FrTensor BD(m * p * nbits, cpu_data_BD);

    auto u_bin = random_vec(log_m + log_p + log_nbits);
    auto v_bin = random_vec(log_m + log_p + log_nbits);

    timer.start();
    auto bin_proof = binary_sumcheck(BD, u_bin, v_bin);
    timer.stop();
    cout << timer.getTotalTime() << endl;
    timer.reset();
    


	delete[] cpu_data_A;
    delete[] cpu_data_B;
    delete[] cpu_data_C;
    delete[] cpu_data_D;
    delete[] cpu_data_BD;
    cout << "Current CUDA status: " << hipGetLastError() << endl;
	return 0;
}
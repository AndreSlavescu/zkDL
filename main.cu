#include "bls12-381.cuh"
#include <iostream>

using namespace std;

typedef blstrs__scalar__Scalar Fr;

int main()
{
  uint size = 1 << 10;
  Fr* gpu_data = nullptr;

  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc((void **)&gpu_data, sizeof(Fr) * size));
  
  cout << gpu_data << endl;

  return 0;
}